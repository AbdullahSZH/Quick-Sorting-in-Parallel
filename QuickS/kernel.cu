#include "hip/hip_runtime.h"
﻿#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>
#include <stdlib.h>

#define STACK_SIZE 32
#define SHARED_SIZE 256

// Swap function to swap two elements
__device__ void swap(int* arr, int i, int j) {
    int temp = arr[i];
    arr[i] = arr[j];
    arr[j] = temp;
}

// Partition function (moved outside of the kernel)
__device__ int partition(int* arr, int low, int high) {
    int pivot = arr[high];
    int i = low - 1;
    for (int j = low; j <= high - 1; j++) {
        if (arr[j] <= pivot) {
            i++;
            swap(arr, i, j);
        }
    }
    swap(arr, i + 1, high);
    return (i + 1);
}

// Parallel quicksort kernel function
__global__ void parallelQuicksort(int* arr, int size) {
    __shared__ int sharedArr[SHARED_SIZE];

    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < size) {
        int low = 0;
        int high = size - 1;
        int stack[STACK_SIZE * 2];
        int top = -1;

        stack[++top] = low;
        stack[++top] = high;

        while (top >= 0) {
            high = stack[top--];
            low = stack[top--];

            int pivot = partition(arr, low, high);

            if (pivot - 1 > low && pivot - 1 <= high) {
                stack[++top] = low;
                stack[++top] = pivot - 1;
            }

            if (pivot + 1 >= low && pivot + 1 < high) {
                stack[++top] = pivot + 1;
                stack[++top] = high;
            }
        }
    }

    // Ensure all threads have finished sorting
    __syncthreads();

    // Each thread loads a block of elements into shared memory
    int sharedIndex = threadIdx.x;
    while (sharedIndex < size) {
        sharedArr[sharedIndex] = arr[sharedIndex];
        sharedIndex += blockDim.x;
    }

    // Ensure all shared memory is loaded
    __syncthreads();

    // Sort the block of elements in shared memory
    for (int i = 0; i < size - 1; i++) {
        for (int j = i + 1; j < size; j++) {
            if (sharedArr[i] > sharedArr[j]) {
                swap(sharedArr, i, j);
            }
        }
    }

    // Ensure all threads have finished sorting
    __syncthreads();

    // Each thread writes back the sorted elements from shared memory to global memory
    sharedIndex = threadIdx.x;
    while (sharedIndex < size) {
        arr[sharedIndex] = sharedArr[sharedIndex];
        sharedIndex += blockDim.x;
    }
}

// Host function to invoke the parallel quicksort kernel
void sortArray(int* arr, int size) {
    int* d_arr;

    hipMalloc((void**)&d_arr, size * sizeof(int));
    hipMemcpy(d_arr, arr, size * sizeof(int), hipMemcpyHostToDevice);

    int numThreadsPerBlock = SHARED_SIZE;
    int numBlocks = (size + numThreadsPerBlock - 1) / numThreadsPerBlock;

    printf("The number of blocks is %d\n", numBlocks);

    parallelQuicksort << <numBlocks, numThreadsPerBlock >> > (d_arr, size);

    hipMemcpy(arr, d_arr, size * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_arr);
}

// Helper function to print the array
void printArray(int* arr, int size) {
    for (int i = 0; i < size; i++) {
        printf("%d ", arr[i]);
    }
    printf("\n");
}

int main() {
    int arr[] = { 7, 2, 1, 6, 8, 5, 3, 4 };
    int size = sizeof(arr) / sizeof(arr[0]);

    printf("Original array: ");
    printArray(arr, size);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

    
    sortArray(arr, size);

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);

    printf("Execution time: %.4f ms\n", elapsedTime);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    printf("Sorted array: ");
    printArray(arr, size);

    return 0;
}